#include "common/book.h"



int main( void ) {
	hipDeviceProp_t prop;
	int dev;

	//int count;
	//HANDLE_ERROR( hipGetDeviceCount( &count ) );

	HANDLE_ERROR( hipGetDevice( &dev ) );
	printf("ID of current CUDA device: %d\n", dev );

	memset( &prop , 0, sizeof(hipDeviceProp_t) );

	prop.major = 1;
	prop.minor = 3;

	HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
	printf("ID of CUDA device closest to revision 1.3: %d\n", dev);
	HANDLE_ERROR( hipSetDevice(dev) );

	printf("Number of blocks allowed along each dimension of a grid: %d\n", prop.maxGridSize[0]);
	/*
	for( int i = 0; i < count; i++ ) {
		HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
		printf( "Name: %s\n", prop.name);
		printf( "CUDA Compute Capability: %d.%d\n", prop.major, prop.minor);
	}
	*/
}
